#include "hip/hip_runtime.h"
#include <cstdlib>
#include <iostream>
#include <fstream>
#include <algorithm>
#include <boost/timer/timer.hpp>
#include <boost/algorithm/string.hpp>

#include "mblas/matrix.h"
#include "model.h"
#include "encoder.h"
#include "decoder.h"
#include "vocab.h"

#include "states.h"

using namespace mblas;

int main(int argc, char** argv) {
  size_t device = 0;
  
  if(argc > 1) {
    if(std::string(argv[1]) == "1")
      device = 1;
    else if(std::string(argv[1]) == "2")
      device = 2;
  }
  
  std::cerr << device << std::endl;
  hipSetDevice(device);
  
  //std::string source = "you know , one of the intense pleasures of travel and one of the delights of ethnographic research is the opportunity to live amongst those who have not forgotten the old ways , who still feel their past in the wind , touch it in stones polished by rain , taste it in the bitter leaves of plants .";
  //std::string target = "wissen sie , ein intensives vergnügen reisen und die freuden der ethnographischen forschung ist die gelegenheit , unter denen zu leben , die alten möglichkeiten nicht vergessen , die noch ihre vergangenheit in den wind fühlen , berühren sie steine polierten durch regen , der bitteren geschmack aus pflanzen .";
  
  std::string source = "just to know that jaguar shamans still journey beyond the milky way , or the myths of the inuit elders still resonate with meaning , or that in the himalaya , the buddhists still pursue the breath of the dharma , is to really remember the central revelation of anthropology , and that is the idea that the world in which we live does not exist in some absolute sense , but is just one model of reality , the consequence of one particular set of adaptive choices that our lineage made , albeit successfully , many generations ago .";
  std::string target = "nur um zu wissen , dass jaguar schamanen noch jenseits der milchstraße reise , oder die mythen der inuit elders noch mit sinn oder dröhnen im himalaya , die buddhisten immer noch den atem des dharma verfolgen , ist wirklich an die zentrale offenbarung der anthropologie , und das ist die idee , dass wir in der welt leben nicht existieren , sondern einen absoluten spüre der realität , nur ein modell aus der reihe von adaptiven entscheidungen , die man vor allem unserer abstammung , wenn auch erfolgreich , aus vielen generationen zurückliegt .";
  
  std::cerr << "Loading model" << std::endl;
  Weights weights("/home/marcinj/Badania/best_nmt/search_model.npz", device);
  Vocab svcb("/home/marcinj/Badania/best_nmt/vocab/en_de.en.txt");
  Vocab tvcb("/home/marcinj/Badania/best_nmt/vocab/en_de.de.txt");
  
  std::cerr << "Creating encoder" << std::endl;
  Encoder encoder(weights);
  std::cerr << "Creating decoder" << std::endl;
  Decoder decoder(weights);
  
  std::vector<std::string> sourceSplit;
  boost::split(sourceSplit, source, boost::is_any_of(" "),
               boost::token_compress_on);
    
  std::cerr << "Source: " << std::endl;
  std::vector<size_t> sWords(sourceSplit.size());
  std::transform(sourceSplit.begin(), sourceSplit.end(), sWords.begin(),
                 [&](const std::string& w) { std::cerr << svcb[w] << ", "; return svcb[w]; });
  sWords.push_back(svcb["</s>"]);
  std::cerr << svcb["</s>"] << std::endl;
  
  typedef std::vector<size_t> Batch;
  
  std::vector<std::string> targetSplit;
  boost::split(targetSplit, target, boost::is_any_of(" "),
               boost::token_compress_on);
    
  std::cerr << "Target: " << std::endl;
  size_t bs = 1000;
  std::vector<std::vector<size_t>> tWordsBatch(targetSplit.size());
  std::transform(targetSplit.begin(), targetSplit.end(), tWordsBatch.begin(),
                 [&](const std::string& w) { std::cerr << tvcb[w] << ", "; return Batch(bs, tvcb[w]); });
  tWordsBatch.push_back(Batch(bs, tvcb["</s>"]));
  std::cerr << tvcb["</s>"] << std::endl;
  
  
  mblas::Matrix SourceContext;
  encoder.GetContext(sWords, SourceContext);
  
  mblas::Matrix PrevState;
  mblas::Matrix PrevEmbedding;
  
  mblas::Matrix AlignedSourceContext;
  mblas::Matrix Probs;
  
  mblas::Matrix State;
  mblas::Matrix Embedding;
  
  std::cerr << "Testing" << std::endl;
  boost::timer::auto_cpu_timer timer;
  size_t batchSize = tWordsBatch[0].size();
  
  for(size_t i = 0; i < 1; ++i) {
    decoder.EmptyState(PrevState, SourceContext, batchSize);      
    decoder.EmptyEmbedding(PrevEmbedding, batchSize);
    
    float sum = 0;
    for(auto w : tWordsBatch) {
      decoder.GetProbs(Probs, AlignedSourceContext,
                       PrevState, PrevEmbedding, SourceContext);
      
      for(size_t i = 0; i < 1; ++i) {
        float p = Probs(i, w[i]);
        std:: cout << log(p) << " ";
        if(i == 0) {  
          sum += log(p);
        }
      }
      std::cout << std::endl;
      
      decoder.Lookup(Embedding, w);
      decoder.GetNextState(State, Embedding,
                           PrevState, AlignedSourceContext);
      
      mblas::Swap(State, PrevState);
      mblas::Swap(Embedding, PrevEmbedding);
    }
    std::cerr << sum << std::endl;
  }
}
