#include "hip/hip_runtime.h"
#include <cstdlib>
#include <iostream>
#include <fstream>
#include <algorithm>
#include <boost/timer/timer.hpp>
#include <boost/algorithm/string.hpp>

#include "mblas/matrix.h"
#include "dl4mt.h"
#include "vocab.h"

using namespace mblas;

int main(int argc, char** argv) {
  size_t device = 0;
  
  if(argc > 1) {
    if(std::string(argv[1]) == "1")
      device = 1;
    else if(std::string(argv[1]) == "2")
      device = 2;
  }
  
  std::cerr << device << std::endl;
  hipSetDevice(device);
  
  std::string source = "thank you .";
  std::string target = "vielen Dank .";
  
  std::cerr << "Loading model" << std::endl;
  Weights weights("testmodel/model.npz", device);
  
  Vocab svcb("testmodel/vocab.en.txt");
  Vocab tvcb("testmodel/vocab.de.txt");
  
  std::cerr << "Creating encoder" << std::endl;
  Encoder encoder(weights);
  
  /*
  std::cerr << "Creating decoder" << std::endl;
  Decoder decoder(weights);
  */
  
  std::vector<std::string> sourceSplit;
  boost::split(sourceSplit, source, boost::is_any_of(" "),
               boost::token_compress_on);
    
  std::cerr << "Source: " << std::endl;
  std::vector<size_t> sWords(sourceSplit.size());
  std::transform(sourceSplit.begin(), sourceSplit.end(), sWords.begin(),
                 [&](const std::string& w) { std::cerr << svcb[w] << ", "; return svcb[w]; });
  sWords.push_back(svcb["</s>"]);
  std::cerr << svcb["</s>"] << std::endl;
  
  typedef std::vector<size_t> Batch;
  
  std::vector<std::string> targetSplit;
  boost::split(targetSplit, target, boost::is_any_of(" "),
               boost::token_compress_on);
    
  std::cerr << "Target: " << std::endl;
  size_t bs = 1000;
  std::vector<std::vector<size_t>> tWordsBatch(targetSplit.size());
  std::transform(targetSplit.begin(), targetSplit.end(), tWordsBatch.begin(),
                 [&](const std::string& w) { std::cerr << tvcb[w] << ", "; return Batch(bs, tvcb[w]); });
  tWordsBatch.push_back(Batch(bs, tvcb["</s>"]));
  std::cerr << tvcb["</s>"] << std::endl;

  mblas::Matrix SourceContext;
  encoder.GetContext(sWords, SourceContext);
  mblas::debug1(SourceContext);

  /*
  mblas::Matrix PrevState;
  mblas::Matrix PrevEmbedding;

  mblas::Matrix AlignedSourceContext;
  mblas::Matrix Probs;

  mblas::Matrix State;
  mblas::Matrix Embedding;

  std::cerr << "Testing" << std::endl;
  boost::timer::auto_cpu_timer timer;
  size_t batchSize = tWordsBatch[0].size();

  for(size_t i = 0; i < 1; ++i) {
    decoder.EmptyState(PrevState, SourceContext, batchSize);
    decoder.EmptyEmbedding(PrevEmbedding, batchSize);
    
    float sum = 0;
    for(auto w : tWordsBatch) {      
      decoder.GetProbs(Probs, AlignedSourceContext,
                       PrevState, PrevEmbedding, SourceContext);

      for(size_t i = 0; i < 1; ++i) {
        float p = Probs(i, w[i]);
        std:: cerr << log(p) << " ";
        if(i == 0) {
          sum += log(p);
        }
      }
      std::cerr << std::endl;

      decoder.Lookup(Embedding, w);
      decoder.GetNextState(State, Embedding,
                           PrevState, AlignedSourceContext);

      mblas::Swap(State, PrevState);
      mblas::Swap(Embedding, PrevEmbedding);
    }
    std::cout << std::endl;
    std::cerr << sum << std::endl;
  }
  
  */
}
