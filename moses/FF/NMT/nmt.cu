#include "hip/hip_runtime.h"
#include <cstdlib>
#include <iostream>
#include <algorithm>
#include <string>
#include <boost/timer/timer.hpp>

#include "nmt.h"
#include "mblas/matrix.h"
#include "model.h"
#include "encoder.h"
#include "decoder.h"
#include "vocab.h"

using namespace mblas;

NMT::NMT(const std::string& model,
         const std::string& src,
         const std::string& trg)
  : w_(new Weights(model)), src_(new Vocab(src)), trg_(new Vocab(trg)),
    encoder_(new Encoder(*w_)), decoder_(new Decoder(*w_))
  {}

void NMT::CalcSourceContext(const std::vector<std::string>& s) {
  std::vector<size_t> words(s.size());
  std::transform(s.begin(), s.end(), words.begin(),
                 [&](const std::string& w) { return (*src_)[w]; });
  words.push_back((*src_)["</s>"]);
  
  SourceContext.reset(new Matrix());
  Matrix& sc = *boost::static_pointer_cast<Matrix>(SourceContext);
  encoder_->GetContext(words, sc);
  debug1(sc);
}

void NMT::MakeStep(
  const std::vector<std::string>& nextWords,
  const std::vector<std::string>& lastWords,
  std::vector<WhichState>& inputStates,
  std::vector<double>& logProbs,
  std::vector<WhichState>& nextStates,
  std::vector<bool>& unks) {
  
  Matrix& sourceContext = *boost::static_pointer_cast<Matrix>(SourceContext);
  
  Matrix lastEmbeddings;
  FillEmbeddings(lastEmbeddings, lastWords, decoder_);
  
  std::vector<size_t> ids;
  Matrix nextEmbeddings;
  FillEmbeddings(nextEmbeddings, nextWords, ids, unks, decoder_);
  
  Matrix prevStates;
  ConstructPrevStates(prevStates, inputStates, states_);
  
  Matrix probs;
  Matrix alignedSourceContex;
  decoder_.GetProbs(probs, alignedSourceContext,
                    prevStates, lastEmbeddings, sourceContext);  

  for(size_t i = 0; i < ids.size(); ++j) {
    float p = Probs(i, ids[i]);
    logProbs.push_back(log(p));
  }
                    
  states_.push_back(new Matrix());
  Matrix& nextStates = *boost::static_pointer_cast<Matrix>(states.back());
  decoder_.GetNextState(nextStates, nextEmbeddings,
                        prevStates, AlignedSourceContext);
  
  size_t current = 0;
  for(size_t i = 0; i < nextStates.Rows(); ++i) {
    nextStates.push_back(Which(states_.size() - 1, i));
  }
}
