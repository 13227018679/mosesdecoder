#include "hip/hip_runtime.h"
#include <cstdlib>
#include <iostream>
#include <algorithm>
#include <string>
#include <boost/timer/timer.hpp>

#include "nmt.h"
#include "mblas/matrix.h"
#include "model.h"
#include "encoder.h"
#include "decoder.h"
#include "vocab.h"

using namespace mblas;

NMT::NMT(const boost::shared_ptr<Weights> model,
         const boost::shared_ptr<Vocab> src,
         const boost::shared_ptr<Vocab> trg)
  : w_(model), src_(src), trg_(trg),
    encoder_(new Encoder(*w_)), decoder_(new Decoder(*w_))
  { }

size_t NMT::GetDevices() {
  int num_gpus = 0;   // number of CUDA GPUs
  hipGetDeviceCount(&num_gpus);
  std::cerr << "Number of CUDA devices: " << num_gpus << std::endl;

  for (int i = 0; i < num_gpus; i++) {
      hipDeviceProp_t dprop;
      hipGetDeviceProperties(&dprop, i);
      std::cerr << i << ": " << dprop.name << std::endl;
  }
  return (size_t)num_gpus;
}

void NMT::SetDevice() {
  hipSetDevice(w_->GetDevice());
}

void NMT::ClearStates() { 
  std::vector<boost::shared_ptr<mblas::BaseMatrix> > temp;
  states_.swap(temp);
}

boost::shared_ptr<Weights> NMT::NewModel(const std::string& path, size_t device) {
  boost::shared_ptr<Weights> weights(new Weights(path, device));
  return weights;
}

boost::shared_ptr<Vocab> NMT::NewVocab(const std::string& path) {
  boost::shared_ptr<Vocab> vocab(new Vocab(path));
  return vocab;
}

void NMT::CalcSourceContext(const std::vector<std::string>& s) {  
  std::vector<size_t> words(s.size());
  std::transform(s.begin(), s.end(), words.begin(),
                 [&](const std::string& w) { return (*src_)[w]; });
  words.push_back((*src_)["</s>"]);
  
  SourceContext.reset(new Matrix());
  Matrix& sc = *boost::static_pointer_cast<Matrix>(SourceContext);
  encoder_->GetContext(words, sc);
  
  // Put empty decoder state into state cache
  states_.emplace_back(new Matrix());
  Matrix& firstStates = *boost::static_pointer_cast<Matrix>(states_.back());
  decoder_->EmptyState(firstStates, sc, 1);
}

void ConstructPrevStates(Matrix& States,
                         const std::vector<WhichState>& inputStates,
                         const std::vector<boost::shared_ptr<mblas::BaseMatrix> >& states) {
  for(auto w: inputStates) {
    //std::cerr << w.stateId << " " << w.rowNo << std::endl;
    Matrix& State = *boost::static_pointer_cast<Matrix>(states[w.stateId]);
    // @TODO: do that with preallocation
    AppendRow(States, State, w.rowNo);
  }
}

void NMT::MakeStep(
  const std::vector<std::string>& nextWords,
  const std::vector<std::string>& lastWords,
  std::vector<WhichState>& inputStates,
  std::vector<double>& logProbs,
  std::vector<WhichState>& outputStates,
  std::vector<bool>& unks) {
  
  Matrix& sourceContext = *boost::static_pointer_cast<Matrix>(SourceContext);
  
  Matrix lastEmbeddings;
  if(states_.size() > 1) {
    // Not the first word
    std::vector<size_t> lastIds(lastWords.size());
    std::transform(lastWords.begin(), lastWords.end(), lastIds.begin(),
                   [&](const std::string& w) { return (*trg_)[w]; });
    decoder_->Lookup(lastEmbeddings, lastIds);
  }
  else {
    // Only empty state in state cache, so this is the first word
    decoder_->EmptyEmbedding(lastEmbeddings, lastWords.size());
  }
  
  Matrix nextEmbeddings;
  std::vector<size_t> nextIds(nextWords.size());
  std::transform(nextWords.begin(), nextWords.end(), nextIds.begin(),
                 [&](const std::string& w) { return (*trg_)[w]; });
  decoder_->Lookup(nextEmbeddings, nextIds);
  for(auto id : nextIds) {
    if(id == 1)
      unks.push_back(true);
    else
      unks.push_back(false);
  }
  
  Matrix prevStates;
  ConstructPrevStates(prevStates, inputStates, states_);

  Matrix probs;
  Matrix alignedSourceContext;
  decoder_->GetProbs(probs, alignedSourceContext,
                     prevStates, lastEmbeddings, sourceContext);  
  
  for(size_t i = 0; i < nextIds.size(); ++i) {
    float p = probs(i, nextIds[i]);
    logProbs.push_back(log(p));
  }
                    
  states_.emplace_back(new Matrix());
  Matrix& nextStates = *boost::static_pointer_cast<Matrix>(states_.back());
  decoder_->GetNextState(nextStates, nextEmbeddings,
                        prevStates, alignedSourceContext);
  
  size_t current = states_.size() - 1;
  for(size_t i = 0; i < nextStates.Rows(); ++i) {
    outputStates.push_back(WhichState(current, i));
  }
}
