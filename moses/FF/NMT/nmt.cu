#include "hip/hip_runtime.h"
#include <cstdlib>
#include <iostream>
#include <algorithm>
#include <string>
#include <boost/timer/timer.hpp>

#include "nmt.h"
#include "mblas/matrix.h"
#include "model.h"
#include "encoder.h"
#include "decoder.h"
#include "vocab.h"
#include "states.h"

using namespace mblas;

NMT::NMT(const boost::shared_ptr<Weights> model,
         const boost::shared_ptr<Vocab> src,
         const boost::shared_ptr<Vocab> trg)
  : w_(model), src_(src), trg_(trg),
    encoder_(new Encoder(*w_)), decoder_(new Decoder(*w_)),
    states_(new States()), firstWord_(true)
  { }

size_t NMT::GetDevices(size_t maxDevices) {
  int num_gpus = 0;   // number of CUDA GPUs
  hipGetDeviceCount(&num_gpus);
  std::cerr << "Number of CUDA devices: " << num_gpus << std::endl;
  
  for (int i = 0; i < num_gpus; i++) {
      hipDeviceProp_t dprop;
      hipGetDeviceProperties(&dprop, i);
      std::cerr << i << ": " << dprop.name << std::endl;
  }
  return (size_t)std::min(num_gpus, (int)maxDevices);
}

void NMT::SetDevice() {
  hipSetDevice(w_->GetDevice());
}

size_t NMT::GetDevice() {
  return w_->GetDevice();
}

void NMT::ClearStates() { 
  states_->Clear();
}

boost::shared_ptr<Weights> NMT::NewModel(const std::string& path, size_t device) {
  hipSetDevice(device);
  boost::shared_ptr<Weights> weights(new Weights(path, device));
  return weights;
}

boost::shared_ptr<Vocab> NMT::NewVocab(const std::string& path) {
  boost::shared_ptr<Vocab> vocab(new Vocab(path));
  return vocab;
}

void NMT::CalcSourceContext(const std::vector<std::string>& s) {  
  std::vector<size_t> words(s.size());
  std::transform(s.begin(), s.end(), words.begin(),
                 [&](const std::string& w) { return (*src_)[w]; });
  words.push_back((*src_)["</s>"]);
  
  SourceContext_.reset(new Matrix());
  Matrix& SC = *boost::static_pointer_cast<Matrix>(SourceContext_);
  encoder_->GetContext(words, SC);
}

StateInfoPtr NMT::EmptyState() {
  Matrix& SC = *boost::static_pointer_cast<Matrix>(SourceContext_);
  Matrix Empty;
  decoder_->EmptyState(Empty, SC, 1);
  std::vector<StateInfoPtr> infos;
  states_->SaveStates(infos, Empty);
  return infos.back();
}

void NMT::MakeStep(
  const std::vector<std::string>& nextWords,
  const std::vector<std::string>& lastWords,
  std::vector<StateInfoPtr>& inputStates,
  std::vector<double>& logProbs,
  std::vector<StateInfoPtr>& outputStates,
  std::vector<bool>& unks) {
  
  Matrix& sourceContext = *boost::static_pointer_cast<Matrix>(SourceContext_);
  
  Matrix lastEmbeddings;
  if(firstWord_) {
    firstWord_ = false;
    // Only empty state in state cache, so this is the first word
    decoder_->EmptyEmbedding(lastEmbeddings, lastWords.size());
  }
  else {
    // Not the first word
    std::vector<size_t> lastIds(lastWords.size());
    std::transform(lastWords.begin(), lastWords.end(), lastIds.begin(),
                   [&](const std::string& w) { return (*trg_)[w]; });
    decoder_->Lookup(lastEmbeddings, lastIds);
  }
  
  Matrix nextEmbeddings;
  std::vector<size_t> nextIds(nextWords.size());
  std::transform(nextWords.begin(), nextWords.end(), nextIds.begin(),
                 [&](const std::string& w) { return (*trg_)[w]; });
  decoder_->Lookup(nextEmbeddings, nextIds);
  for(auto id : nextIds) {
    if(id != 1)
      unks.push_back(true);
    else
      unks.push_back(false);
  }
  
  Matrix prevStates;
  states_->ConstructStates(prevStates, inputStates);

  Matrix probs;
  Matrix alignedSourceContext;
  decoder_->GetProbs(probs, alignedSourceContext,
                     prevStates, lastEmbeddings, sourceContext);  
  
  for(size_t i = 0; i < nextIds.size(); ++i) {
    float p = probs(i, nextIds[i]);
    logProbs.push_back(log(p));
  }
                    
  Matrix nextStates;
  decoder_->GetNextState(nextStates, nextEmbeddings,
                        prevStates, alignedSourceContext);
  states_->SaveStates(outputStates, nextStates);
  
}
