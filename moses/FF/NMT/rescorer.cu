#include "hip/hip_runtime.h"

#include <cstdlib>
#include <iostream>
#include <string>
#include <algorithm>
#include <boost/timer/timer.hpp>
#include <boost/program_options/options_description.hpp>
#include <boost/program_options/parsers.hpp>
#include <boost/program_options/variables_map.hpp>
#include <boost/lexical_cast.hpp>

#include "mblas/matrix.h"
#include "model.h"
#include "encoder.h"
#include "decoder.h"
#include "vocab.h"

#include "states.h"

void ProgramOptions(int argc, char *argv[],
    std::string& modelPath,
    std::string& svPath,
    std::string& tvPath,
    std::string& corpusPath,
    std::string& nbestPath,
    std::string& fname,
    size_t& maxBatchSize,
    size_t& device) {
  bool help = false;

  namespace po = boost::program_options;
  po::options_description cmdline_options("Allowed options");
  cmdline_options.add_options()
    ("device,d", po::value(&device)->default_value(0),
     "CUDA Device")
    ("batch,b", po::value(&maxBatchSize)->default_value(1000),
     "Max batch size")
    ("model,m", po::value(&modelPath)->required(),
     "Path to a model")
    ("source,s", po::value(&svPath)->required(),
     "Path to a source vocab file.")
    ("target,t", po::value(&tvPath)->required(),
     "Path to a target vocab file.")
    ("input,i", po::value(&corpusPath)->required(),
     "Path to the input of the nbest file.")
    ("n-best,n", po::value(&nbestPath)->required(),
     "Path to an nbest file.")
    ("feature-name,f", po::value(&fname)->default_value("NMT0"),
     "Feature name")
    ("help,h", po::value(&help)->zero_tokens()->default_value(false),
     "Print this help message and exit.")
  ;
  po::variables_map vm;
  try {
    po::store(po::command_line_parser(argc, argv).
              options(cmdline_options).run(), vm);
    po::notify(vm);
  } catch (std::exception& e) {
    std::cout << "Error: " << e.what() << std::endl << std::endl;

    std::cout << "Usage: " + std::string(argv[0]) +  " [options]" << std::endl;
    std::cout << cmdline_options << std::endl;
    exit(0);
  }

  if (help) {
    std::cout << "Usage: " + std::string(argv[0]) +  " [options]" << std::endl;
    std::cout << cmdline_options << std::endl;
    exit(0);
  }
}

void ParseInputFile(std::string path, std::vector<std::string>& output) {
    std::ifstream file(path);
    output.clear();
    std::string line;
    while (std::getline(file, line).good()) {
      line += " </s>";
      output.push_back(line);
    }
}

std::vector<std::string> Split(std::string& line, std::string del=" ") {
    std::vector<std::string> output;
    size_t pos = 0;
    std::string token;
    while ((pos = line.find(del)) != std::string::npos) {
        token = line.substr(0, pos);
        output.push_back(token);
        line.erase(0, pos + del.size());
    }
    output.push_back(line);
    return output;
}

void ParseNBestFile(std::string path, std::vector<std::vector<std::string>>& output) {
    std::ifstream file(path);
    std::string line;
    while (std::getline(file, line).good()) {
        output.push_back(Split(line, " ||| "));
    }
}

void PrepareBatch(const std::vector<std::vector<size_t>>& input,
                  std::vector<std::vector<size_t>>& output) {

    size_t maxSentenceLength = 0;
    for (auto& sentence: input) {
        maxSentenceLength = max(maxSentenceLength, sentence.size());
    }

    for (size_t i = 0; i < maxSentenceLength; ++i) {
        output.emplace_back(input.size(), 0);
        for (size_t j = 0; j < input.size(); ++j) {
            if (i < input[j].size()) {
                output[i][j] = input[j][i];
            }
        }
    }
}

int main(int argc, char* argv[]) {
  std::string modelPath, svPath, tvPath, corpusPath, nbestPath, fname;

  size_t device;
  size_t maxBatchSize;
  ProgramOptions(argc, argv, modelPath, svPath,tvPath, corpusPath, nbestPath,
                 fname, maxBatchSize, device);
  hipSetDevice(device);
  std::cerr << "Loading model: " << modelPath << std::endl;
  Weights weights(modelPath, device);
  Vocab svcb(svPath);
  Vocab tvcb(tvPath);

  std::vector<std::string> input;
  ParseInputFile(corpusPath, input);

  std::vector<std::vector<std::string>> nbest;
  ParseNBestFile(nbestPath, nbest);

  size_t index = 0;
  size_t nbestIndex = 0;
  for (auto& in: input) {
    Encoder encoder(weights);
    Decoder decoder(weights);

    auto words = Split(in);
    auto sIndexes = svcb.Encode(words);

    std::vector<std::vector<size_t> > sentences2score;
    while (nbestIndex < nbest.size()) {
      sentences2score.clear();
      for (; (nbestIndex < nbest.size()) && (sentences2score.size() < maxBatchSize); ++nbestIndex) {
        if (boost::lexical_cast<size_t>(nbest[nbestIndex][0]) == index) {
          std::string sentence = nbest[nbestIndex][1] + " </s>";
          sentences2score.push_back(tvcb.Encode(Split(sentence)));
        }
        else {
          break;
        }

      }
      if (sentences2score.size() == 0 ) {
        index = boost::lexical_cast<size_t>(nbest[nbestIndex][0]);
        continue;
      }

      std::vector<std::vector<size_t>> batch;
      PrepareBatch(sentences2score, batch);

      if(index > 0 && index % 5 == 0)
        std::cerr << ".";
      if(index > 0 && index % 100 == 0)
        std::cerr << "[" << index << "]" << std::endl;

      mblas::Matrix SourceContext;
      encoder.GetContext(sIndexes, SourceContext);

      mblas::Matrix PrevState;
      mblas::Matrix PrevEmbedding;

      mblas::Matrix AlignedSourceContext;
      mblas::Matrix Probs;

      mblas::Matrix State;
      mblas::Matrix Embedding;
      size_t batchSize = batch[0].size();

      decoder.EmptyState(PrevState, SourceContext, batchSize);
      decoder.EmptyEmbedding(PrevEmbedding, batchSize);

      std::vector<float> scores(batch[0].size(), 0.0f);
      size_t lengthIndex = 0;
      for (auto& w : batch) {
        decoder.GetProbs(Probs, AlignedSourceContext,
                         PrevState, PrevEmbedding, SourceContext);

        for (size_t j = 0; j < w.size(); ++j) {
          if (batch[lengthIndex][j]) {
            float p = Probs(j, w[j]);
            scores[j] += log(p);
          }
        }

        decoder.Lookup(Embedding, w);
        decoder.GetNextState(State, Embedding,
                             PrevState, AlignedSourceContext);

        mblas::Swap(State, PrevState);
        mblas::Swap(Embedding, PrevEmbedding);
        ++lengthIndex;
      }
      for (size_t j = 0; j < batch[0].size(); ++j) {
        std::cout
          << nbest[nbestIndex - sentences2score.size() + j][0] << " ||| "
          << nbest[nbestIndex - sentences2score.size() + j][1] << " ||| "
          << nbest[nbestIndex - sentences2score.size() + j][2] << " " 
          << fname << "= " << scores[j] << " ||| "
          << nbest[nbestIndex - sentences2score.size() + j][3] << std::endl;
      }
      index = boost::lexical_cast<size_t>(nbest[nbestIndex][0]);
    }
  }
  return 0;
}
