#include "hip/hip_runtime.h"
#include "nbest.h"

#include <algorithm>

#include "utils.h"
#include "vocab.h"

NBest::NBest(
    const std::string& nbestPath,
    const std::string& trgVocabPath)
    : trgVocab_(trgVocabPath) {
  Parse_(nbestPath);
}

void NBest::Parse_(const std::string& path) {
  std::ifstream file(path);

  std::string line;
  size_t lineCount = 0;
  indexes_.push_back(lineCount);

  while (std::getline(file, line).good()) {
    boost::trim(line);
    std::vector<std::string> fields;
    Split(line, fields, " ||| ");
    if (lineCount && (data_.back()[0] != fields[0])) {
      indexes_.push_back(lineCount);
    }
    data_.push_back(fields);
  }
  indexes_.push_back(data_.size());
}


inline std::vector< std::vector< std::string > > NBest::SplitBatch(std::vector<std::string>& batch) const {
  std::vector< std::vector< std::string > > splittedBatch;
  for (auto& sentence : batch) {
    Trim(sentence);
    std::vector<std::string> words;
    Split(sentence, words);
    splittedBatch.push_back(words);
  }
  return splittedBatch;
}

inline Batch NBest::EncodeBatch(const std::vector<std::vector<std::string>>& batch) const {
  Batch encodedBatch;
  for (auto& sentence: batch) {
    encodedBatch.push_back(trgVocab_.Encode(sentence, true));
  }
  return encodedBatch;
}

inline Batch NBest::MaskAndTransposeBatch(const Batch& batch) const {
  size_t maxLength = 0;
  for (auto& sentence: batch) {
    maxLength = max(maxLength, sentence.size());
  }
  Batch masked;
  for (size_t i = 0; i < maxLength; ++i) {
      masked.emplace_back(batch.size(), 0);
      for (size_t j = 0; j < batch.size(); ++j) {
          if (i < batch[j].size()) {
              masked[i][j] = batch[j][i];
          }
      }
  }
  return masked;
}


Batch NBest::ProcessBatch(std::vector<std::string>& batch) const {
  return MaskAndTransposeBatch(EncodeBatch(SplitBatch(batch)));
}

std::vector<Batch> NBest::GetBatches(const size_t index, size_t maxSize) const {
  std::vector<Batch> batches;
  std::vector<std::string> sBatch;
  for (size_t i = indexes_[index]; i <= indexes_[index+1]; ++i) {
    if (sBatch.size() == maxSize || i == indexes_[index+1]) {
      batches.push_back(ProcessBatch(sBatch));
      sBatch.clear();
      if (i == indexes_[index+1]) {
        break;
      }
    }
    sBatch.push_back(data_[i][1]);
  }
  return batches;
}
