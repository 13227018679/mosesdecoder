#include "hip/hip_runtime.h"
#include "rescorer.h"

#include <cstdlib>
#include <iostream>
#include <string>

#include "encoder.h"
#include "decoder.h"
#include "vocab.h"
#include "nbest.h"
#include "utils.h"

#include "states.h"
#include "encoder.h"
#include "decoder.h"
#include "nbest.h"

Rescorer::Rescorer(
  const std::shared_ptr<Weights> model,
  const std::shared_ptr<NBest> nBest,
  const std::string& featureName)
    : model_(model),
      encoder_(new Encoder(*model)),
      decoder_(new Decoder(*model)),
      featureName_(featureName),
      nbest_(nBest) {
}

std::vector<float> Rescorer::ScoreBatch(
    void* SourceContext,
    const std::vector<std::vector<size_t>>& batch) {
  mblas::Matrix PrevState;
  mblas::Matrix PrevEmbedding;

  mblas::Matrix AlignedSourceContext;
  mblas::Matrix Probs;

  mblas::Matrix State;
  mblas::Matrix Embedding;
  size_t batchSize = batch[0].size();

  decoder_->EmptyState(PrevState, *(mblas::Matrix*)SourceContext, batchSize);
  decoder_->EmptyEmbedding(PrevEmbedding, batchSize);

  std::vector<float> scores(batch[0].size(), 0.0f);
  size_t lengthIndex = 0;
  for (auto& w : batch) {
    decoder_->GetProbs(Probs, AlignedSourceContext,
                     PrevState, PrevEmbedding, *(mblas::Matrix*)SourceContext);

    for (size_t j = 0; j < w.size(); ++j) {
      if (batch[lengthIndex][j]) {
        float p = Probs(j, w[j]);
        scores[j] += log(p);
      }
    }

    decoder_->Lookup(Embedding, w);
    decoder_->GetNextState(State, Embedding,
                         PrevState, AlignedSourceContext);

    mblas::Swap(State, PrevState);
    mblas::Swap(Embedding, PrevEmbedding);
    ++lengthIndex;
  }
  return scores;
}

void Rescorer::Score(const size_t index) {
  auto sIndexes = nbest_->GetEncodedTokens(index);

  mblas::Matrix SourceContext;
  encoder_->GetContext(sIndexes, SourceContext);
  size_t batchIndex = 0;
  for(auto& batch: nbest_->GetBatches(index)) {
    const auto scores = ScoreBatch(&SourceContext, batch);
    for (size_t j = 0; j < batch[0].size(); ++j) {
      std::cerr
        << (*nbest_)[nbest_->GetIndex(index) + batchIndex + j][0] << " ||| "
        << (*nbest_)[nbest_->GetIndex(index) + batchIndex + j][1] << " ||| "
        << (*nbest_)[nbest_->GetIndex(index) + batchIndex + j][2] << " "
        << featureName_ << "= " << scores[j] << " ||| "
        << (*nbest_)[nbest_->GetIndex(index) + batchIndex + j][3]
        << std::endl;
    }
    batchIndex += batch[0].size();
  }
}
