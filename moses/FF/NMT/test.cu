#include "hip/hip_runtime.h"
#include <cstdlib>
#include <iostream>
#include <algorithm>
#include <boost/timer/timer.hpp>

#include "mblas/matrix.h"
#include "model.h"
#include "encoder.h"
#include "decoder.h"
#include "vocab.h"

using namespace mblas;

int main(int argc, char** argv) {
  size_t device = 0;
  
  if(argc > 1)
    device = 1;
  
  hipSetDevice(device);
  //CublasHandle::Init(device);
  
  std::cerr << "Loading model" << std::endl;
  Weights weights("/home/marcinj/Badania/nmt/en_de_1/search_model.npz", device);
  Vocab svcb("/home/marcinj/Badania/nmt/en_de_1/src.vocab.txt");
  Vocab tvcb("/home/marcinj/Badania/nmt/en_de_1/trg.vocab.txt");
  
  std::cerr << "Creating encoder" << std::endl;
  Encoder encoder(weights);
  std::cerr << "Creating decoder" << std::endl;
  Decoder decoder(weights);
  
  std::vector<size_t> sWords = {svcb["this"], svcb["is"], svcb["a"],
                                svcb["little"], svcb["test"], svcb["."],
                                svcb["</s>"]};
  
  //std::vector<std::vector<size_t>> tWordsBatch = {
  //  {  tvcb["das"],     tvcb["dies"],    tvcb["das"]     },
  //  {  tvcb["ist"],     tvcb["war"],     tvcb["ist"]     },
  //  {  tvcb["ein"],     tvcb["ein"],     tvcb["eine"]    },
  //  {  tvcb["kleiner"], tvcb["ganz"],    tvcb["kleine"]  },
  //  {  tvcb["test"],    tvcb["kleiner"], tvcb["frau"]    },
  //  {  tvcb["."],       tvcb["test"],    tvcb["."]       },
  //  {  tvcb["</s>"],    tvcb["."],       tvcb["</s>"]    },
  //  {  0,               tvcb["</s>"],    0               }
  //};
  
  typedef std::vector<size_t> Batch;
  size_t bs = 1;
  std::vector<std::vector<size_t>> tWordsBatch = {
    Batch(bs, tvcb["das"]),
    Batch(bs, tvcb["ist"]),
    Batch(bs, tvcb["ein"]),
    Batch(bs, tvcb["kleiner"]),
    Batch(bs, tvcb["test"]),
    Batch(bs, tvcb["."]),
    Batch(bs, tvcb["</s>"])
  };
    
  mblas::Matrix SourceContext;
  encoder.GetContext(sWords, SourceContext);
  
  mblas::Matrix PrevState;
  mblas::Matrix PrevEmbedding;

  mblas::Matrix AlignedSourceContext;
  mblas::Matrix Probs;
  
  mblas::Matrix State;
  mblas::Matrix Embedding;

  std::cerr << "Testing" << std::endl;
  boost::timer::auto_cpu_timer timer;
  size_t batchSize = tWordsBatch[0].size();

  decoder.EmptyState(PrevState, SourceContext, batchSize);
  decoder.EmptyEmbedding(PrevEmbedding, batchSize);
  
  float sum = 0;
  for(auto w : tWordsBatch) {
  
    decoder.GetProbs(Probs, AlignedSourceContext,
                     PrevState, PrevEmbedding, SourceContext);
    
    for(size_t j = 0; j < 1; ++j) {
      float p = Probs(j, w[j]);
      std::cerr << j << " " << w[j] << " " << log(p) << std::endl;
      sum += log(p);
    }
  
    decoder.Lookup(Embedding, w);
    decoder.GetNextState(State, Embedding,
                         PrevState, AlignedSourceContext);
    
    mblas::Swap(State, PrevState);
    mblas::Swap(Embedding, PrevEmbedding);
  }
}
