#include "hip/hip_runtime.h"
#include <cstdlib>
#include <iostream>
#include <algorithm>
#include <boost/timer/timer.hpp>

#include "mblas/matrix.h"
#include "model.h"
#include "encoder.h"
#include "decoder.h"
#include "vocab.h"

using namespace mblas;

int main() {  
  std::cerr << "Loading model" << std::endl;
  Weights weights("../../nmt/en_de_1/search_model.npz");
  Vocab svcb("src.vocab.txt");
  Vocab tvcb("trg.vocab.txt");
  
  std::cerr << "Creating encoder" << std::endl;
  Encoder encoder(weights);
  std::cerr << "Creating decoder" << std::endl;
  Decoder decoder(weights);
  
  std::vector<size_t> sWords = {svcb["this"], svcb["is"], svcb["a"],
                                svcb["little"], svcb["test"], svcb["."],
                                svcb["</s>"]};
  
  std::vector<std::vector<size_t>> tWordsBatch = {
    {  tvcb["das"],     tvcb["dies"],    tvcb["das"]     },
    {  tvcb["ist"],     tvcb["war"],     tvcb["ist"]     },
    {  tvcb["ein"],     tvcb["ein"],     tvcb["eine"]    },
    {  tvcb["kleiner"], tvcb["ganz"],    tvcb["kleine"]  },
    {  tvcb["test"],    tvcb["kleiner"], tvcb["frau"]    },
    {  tvcb["."],       tvcb["test"],    tvcb["."]       },
    {  tvcb["</s>"],    tvcb["."],       tvcb["</s>"]    },
    {  0,               tvcb["</s>"],    0               }
  };
    
  mblas::Matrix SourceContext;
  encoder.GetContext(sWords, SourceContext);
  
  mblas::Matrix PrevState;
  mblas::Matrix PrevEmbedding;

  mblas::Matrix AlignedSourceContext;
  mblas::Matrix Probs;
  
  mblas::Matrix State;
  mblas::Matrix Embedding;

  std::cerr << "Testing" << std::endl;
  boost::timer::auto_cpu_timer timer;
  size_t batchSize = tWordsBatch[0].size();

  decoder.EmptyState(PrevState, SourceContext, batchSize);
  decoder.EmptyEmbedding(PrevEmbedding, batchSize);
  
  float sum = 0;
  for(auto w : tWordsBatch) {
    decoder.GetProbs(Probs, AlignedSourceContext,
                     PrevState, PrevEmbedding, SourceContext);
    
    for(size_t j = 0; j < 3; ++j) {
      float p = Probs(j, w[j]);
      std::cerr << j << " " << w[j] << " " << p << std::endl;
      sum += log(p);
    }
  
    decoder.GetNextState(State, Embedding,
                         w, PrevState, AlignedSourceContext);
    //std::cerr << "State" << std::endl;
    //debug1(State);
    
    mblas::Swap(State, PrevState);
    mblas::Swap(Embedding, PrevEmbedding);
  }
  //std::cerr << sum << std::endl;    
}
