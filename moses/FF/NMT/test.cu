#include "hip/hip_runtime.h"
#include <cstdlib>
#include <iostream>
#include <fstream>
#include <algorithm>
#include <boost/timer/timer.hpp>
#include <boost/algorithm/string.hpp>

#include "mblas/matrix.h"
#include "model.h"
#include "encoder.h"
#include "decoder.h"
#include "vocab.h"

#include "states.h"

using namespace mblas;

int main(int argc, char** argv) {
  size_t device = 0;
  
  if(argc > 1)
    device = 1;
  
  hipSetDevice(device);
  
  std::string source = "you know , one of the intense pleasures of travel and one of the delights of ethnographic research is the opportunity to live amongst those who have not forgotten the old ways , who still feel their past in the wind , touch it in stones polished by rain , taste it in the bitter leaves of plants .";
  
  std::string target = "wissen sie , ein intensives vergnügen reisen und die freuden der ethnographischen forschung ist die möglichkeit , unter denen leben nicht vergessen , die alte art , die sich ihrer vergangenheit noch im wind , berühren sie in steine poliert von regen , geschmack in den bitteren blätter von pflanzen .";
 
  std::cerr << "Loading model" << std::endl;
  Weights weights("/home/marcinj/Badania/best_nmt/search_model.npz", device);
  Vocab svcb("/home/marcinj/Badania/best_nmt/vocab/en_de.en.txt");
  Vocab tvcb("/home/marcinj/Badania/best_nmt/vocab/en_de.de.txt");
  
  std::cerr << "Creating encoder" << std::endl;
  Encoder encoder(weights);
  std::cerr << "Creating decoder" << std::endl;
  Decoder decoder(weights);
  
  std::vector<std::string> sourceSplit;
  boost::split(sourceSplit, source, boost::is_any_of(" "),
               boost::token_compress_on);
    
  std::cerr << "Source: " << std::endl;
  std::vector<size_t> sWords(sourceSplit.size());
  std::transform(sourceSplit.begin(), sourceSplit.end(), sWords.begin(),
                 [&](const std::string& w) { std::cerr << svcb[w] << ", "; return svcb[w]; });
  sWords.push_back(svcb["</s>"]);
  std::cerr << svcb["</s>"] << std::endl;
  
  
  typedef std::vector<size_t> Batch;
  
  std::vector<std::string> targetSplit;
  boost::split(targetSplit, target, boost::is_any_of(" "),
               boost::token_compress_on);
    
  std::cerr << "Target: " << std::endl;
  size_t bs = 100;
  std::vector<std::vector<size_t>> tWordsBatch(targetSplit.size());
  std::transform(targetSplit.begin(), targetSplit.end(), tWordsBatch.begin(),
                 [&](const std::string& w) { std::cerr << tvcb[w] << ", "; return Batch(bs, tvcb[w]); });
  tWordsBatch.push_back(Batch(bs, tvcb["</s>"]));
  std::cerr << tvcb["</s>"] << std::endl;
  
  
  mblas::Matrix SourceContext;
  encoder.GetContext(sWords, SourceContext);
  
  mblas::Matrix PrevState;
  mblas::Matrix PrevEmbedding;
  
  mblas::Matrix AlignedSourceContext;
  mblas::Matrix Probs;
  
  mblas::Matrix State;
  mblas::Matrix Embedding;
  
  std::cerr << "Testing" << std::endl;
  boost::timer::auto_cpu_timer timer;
  size_t batchSize = tWordsBatch[0].size();
  
  for(size_t i = 0; i < 1; ++i) {
    decoder.EmptyState(PrevState, SourceContext, batchSize);
    decoder.EmptyEmbedding(PrevEmbedding, batchSize);
    
    float sum = 0;
    for(auto w : tWordsBatch) {
    
      decoder.GetProbs(Probs, AlignedSourceContext,
                       PrevState, PrevEmbedding, SourceContext);
      
      
      float p = Probs(0, w[0]);
      sum += log(p);
      
      decoder.Lookup(Embedding, w);
      decoder.GetNextState(State, Embedding,
                           PrevState, AlignedSourceContext);
      
      mblas::Swap(State, PrevState);
      mblas::Swap(Embedding, PrevEmbedding);
    }
    
    std::cerr << sum << std::endl;
    
  }
}
