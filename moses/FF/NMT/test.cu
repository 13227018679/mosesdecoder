#include <cstdlib>
#include <iostream>
#include <algorithm>
#include <boost/timer/timer.hpp>

#include "mblas/matrix.h"
#include "model.h"
#include "encoder.h"
#include "decoder.h"
#include "vocab.h"

#include "states.h"

using namespace mblas;

int main(int argc, char** argv) {
  size_t device = 0;
  
  if(argc > 1)
    device = 1;
  
  hipSetDevice(device);
  
  std::cerr << "Loading model" << std::endl;
  Weights weights("/home/marcinj/Badania/nmt/en_de_1/search_model.npz", device);
  Vocab svcb("/home/marcinj/Badania/nmt/en_de_1/src.vocab.txt");
  Vocab tvcb("/home/marcinj/Badania/nmt/en_de_1/trg.vocab.txt");
  
  std::cerr << "Creating encoder" << std::endl;
  Encoder encoder(weights);
  std::cerr << "Creating decoder" << std::endl;
  Decoder decoder(weights);
  
  std::vector<size_t> sWords = {svcb["this"], svcb["is"], svcb["a"],
                                svcb["little"], svcb["test"], svcb["."],
                                svcb["</s>"]};
  
  //std::vector<std::vector<size_t>> tWordsBatch = {
  //  {  tvcb["das"],     tvcb["dies"],    tvcb["das"]     },
  //  {  tvcb["ist"],     tvcb["war"],     tvcb["ist"]     },
  //  {  tvcb["ein"],     tvcb["ein"],     tvcb["eine"]    },
  //  {  tvcb["kleiner"], tvcb["ganz"],    tvcb["kleine"]  },
  //  {  tvcb["test"],    tvcb["kleiner"], tvcb["frau"]    },
  //  {  tvcb["."],       tvcb["test"],    tvcb["."]       },
  //  {  tvcb["</s>"],    tvcb["."],       tvcb["</s>"]    },
  //  {  0,               tvcb["</s>"],    0               }
  //};
  
  typedef std::vector<size_t> Batch;
  size_t bs = 500;
  std::vector<std::vector<size_t>> tWordsBatch = {
    Batch(bs, tvcb["das"]),
    Batch(bs, tvcb["ist"]),
    Batch(bs, tvcb["ein"]),
    Batch(bs, tvcb["kleiner"]),
    Batch(bs, tvcb["test"]),
    Batch(bs, tvcb["."]),
    Batch(bs, tvcb["</s>"])
  };
  
  std::vector<size_t> filter = {
    tvcb["das"], tvcb["ist"], tvcb["ein"], tvcb["kleiner"], tvcb["test"],
    tvcb["."], tvcb["</s>"], 0, tvcb["dies"], tvcb["ist"], tvcb["war"],        
    tvcb["eine"], tvcb["ganz"], tvcb["kleine"], tvcb["frau"], 
  };
    
  decoder.Filter(filter); // Limit to allowed vocabulary
    
  mblas::Matrix SourceContext;
  encoder.GetContext(sWords, SourceContext);
  
  mblas::Matrix PrevState;
  mblas::Matrix PrevEmbedding;
  
  mblas::Matrix AlignedSourceContext;
  mblas::Matrix Probs;
  
  mblas::Matrix State;
  mblas::Matrix Embedding;
  
  std::cerr << "Testing" << std::endl;
  boost::timer::auto_cpu_timer timer;
  size_t batchSize = tWordsBatch[0].size();
  
  for(size_t i = 0; i < 10; ++i) {
    decoder.EmptyState(PrevState, SourceContext, batchSize);
    decoder.EmptyEmbedding(PrevEmbedding, batchSize);
    
    size_t k = 0;
    for(auto w : tWordsBatch) {
    
      decoder.GetProbs(Probs, AlignedSourceContext,
                       PrevState, PrevEmbedding, SourceContext);
      
      
      float p = Probs(0, k);
      std::cerr << k << " " << filter[k++] << " " << p << std::endl;
      
      decoder.Lookup(Embedding, w);
      decoder.GetNextState(State, Embedding,
                           PrevState, AlignedSourceContext);
      
      mblas::Swap(State, PrevState);
      mblas::Swap(Embedding, PrevEmbedding);
    }
  }
}
