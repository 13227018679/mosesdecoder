#include <cstdlib>
#include <iostream>
#include <fstream>
#include <algorithm>
#include <boost/timer/timer.hpp>
#include <boost/algorithm/string.hpp>

#include <thrust/functional.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/device_ptr.h>
#include <thrust/extrema.h>
#include <thrust/sort.h>
#include <thrust/sequence.h>


#include "mblas/matrix.h"
#include "dl4mt.h"
#include "vocab.h"

#include "states.h"

using namespace mblas;

typedef std::tuple<size_t, size_t, float> Hypothesis;
typedef std::vector<Hypothesis> Beam;

void BestHyps(Beam& bestHyps, const Beam& prevHyps, mblas::Matrix& Probs, const size_t beamSize) {
  mblas::Matrix Costs(Probs.Rows(), 1);
  thrust::host_vector<float> vCosts;
  for(const Hypothesis& h : prevHyps)
    vCosts.push_back(std::get<2>(h));
  thrust::copy(vCosts.begin(), vCosts.end(), Costs.begin());
  //mblas::debug1(Costs);
  
  mblas::BroadcastColumn(Log(_1) + _2, Probs, Costs);
  
  thrust::device_vector<unsigned> keys(Probs.size());
  thrust::sequence(keys.begin(), keys.end());
  
  // Tutaj przydalaby sie funkcja typu partition_n zamiast pelnego sort
  thrust::sort_by_key(Probs.begin(), Probs.end(), keys.begin(), thrust::greater<float>());
  
  thrust::host_vector<unsigned> bestKeys(beamSize);
  thrust::copy_n(keys.begin(), beamSize, bestKeys.begin());
  thrust::host_vector<float> bestCosts(beamSize);
  thrust::copy_n(Probs.begin(), beamSize, bestCosts.begin());
  
  for(size_t i = 0; i < beamSize; i++) {
    size_t wordIndex = bestKeys[i] % Probs.Cols();
    size_t hypIndex  = bestKeys[i] / Probs.Cols();
    float  cost = bestCosts[i];
    //std::cerr << wordIndex << " " << hypIndex << " " << cost << std::endl;
    bestHyps.emplace_back(wordIndex, hypIndex, cost);  
  }
}

int main(int argc, char** argv) {
  size_t device = 0;
  
  if(argc > 1) {
    if(std::string(argv[1]) == "1")
      device = 1;
    else if(std::string(argv[1]) == "2")
      device = 2;
  }
  
  std::cerr << device << std::endl;
  hipSetDevice(device);
  
  Weights weights("testmodel/model.npz", device);
  Vocab svcb("testmodel/vocab.en.txt");
  Vocab tvcb("testmodel/vocab.de.txt");
  
  std::cerr << "Creating encoder" << std::endl;
  Encoder encoder(weights);

  std::cerr << "Creating decoder" << std::endl;
  Decoder decoder(weights);
  
  mblas::Matrix State, NextState, BeamState;
  mblas::Matrix Embeddings, NextEmbeddings;
  mblas::Matrix Probs;
    
  std::string source;
  boost::timer::auto_cpu_timer timer;
  
  while(std::getline(std::cin, source)) {
    std::vector<std::string> sourceSplit;
    boost::split(sourceSplit, source, boost::is_any_of(" "),
                 boost::token_compress_on);
      
    std::vector<size_t> sourceWords(sourceSplit.size());
    std::transform(sourceSplit.begin(), sourceSplit.end(), sourceWords.begin(),
                   [&](const std::string& w) { return svcb[w]; });
    sourceWords.push_back(svcb["</s>"]);
    
    mblas::Matrix SourceContext;
    encoder.GetContext(sourceWords, SourceContext);
  
    size_t beamSize = 10;
    
    decoder.EmptyState(State, SourceContext, 1);
    decoder.EmptyEmbedding(Embeddings, 1);
    
    std::vector<Beam> history;
    
    Beam prevHyps;
    prevHyps.emplace_back(0, 0, 0.0);
    
    do {
      decoder.MakeStep(NextState, Probs, State, Embeddings, SourceContext);
      
      Beam hyps;
      BestHyps(hyps, prevHyps, Probs, beamSize);
      
      std::vector<size_t> nextWords(hyps.size());
      std::transform(hyps.begin(), hyps.end(), nextWords.begin(),
                     [](Hypothesis& h) { return std::get<0>(h); });
      decoder.Lookup(NextEmbeddings, nextWords);
      
      std::vector<size_t> beamStateIds(hyps.size());
      std::transform(hyps.begin(), hyps.end(), beamStateIds.begin(),
                     [](Hypothesis& h) { return std::get<1>(h); });
      mblas::Assemble(BeamState, NextState, beamStateIds);
      
      mblas::Swap(Embeddings, NextEmbeddings);
      mblas::Swap(State, BeamState);
      
      history.push_back(hyps);
      prevHyps.swap(hyps);
      
    } while(std::get<0>(prevHyps[0]) != tvcb["</s>"] && history.size() < sourceWords.size() * 3);
    
    std::vector<size_t> targetWords;
    size_t best = 0;
    for(auto b = history.rbegin(); b != history.rend(); b++) {
      auto& bestHyp = (*b)[best];
      targetWords.push_back(std::get<0>(bestHyp));
      best = std::get<1>(bestHyp);
    }
    
    std::reverse(targetWords.begin(), targetWords.end());
    for(size_t i = 0; i < targetWords.size(); ++i) {
      if(tvcb[targetWords[i]] != "</s>") {
        if(i > 0) {
          std::cout << " ";
        }
        std::cout << tvcb[targetWords[i]];
      }
    }
    std::cout << std::endl;
  }
}